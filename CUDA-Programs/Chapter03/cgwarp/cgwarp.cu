#include "hip/hip_runtime.h"
// Programming in Parallel with CUDA - supporting code by Richard Ansorge 
// copyright 2021 is licensed under CC BY-NC 4.0 for non-commercial use
// This code may be freely changed but please retain an acknowledgement

// cgwarp example 3.3
//
// RTX 2070 
// C:\bin\cgwarp.exe 1234567 28800 256
// warps and subwarps for thread 1234567:
// warp32 rank in tile  7 tile size 32 tile rank    4 tile number   8 net size 256
// warp16 rank in tile  7 tile size 16 tile rank    8 tile number  16 net size 256
// warp8  rank in tile  7 tile size  8 tile rank   16 tile number  32 net size 256
// tile8  rank in tile  7 tile size  8 tile rank    0 tile number   4 net size 32
// tile4  rank in tile  3 tile size  4 tile rank    1 tile number   2 net size 8
//
// // RTX 3080
// C:\bin\cgwarp.exe 1234567 28800 256
// warps and subwarps for thread 1234567:
// warp32 rank in tile  7 tile size 32 tile rank    4 tile number   8 net size 256
// warp16 rank in tile  7 tile size 16 tile rank    8 tile number  16 net size 256
// warp8  rank in tile  7 tile size  8 tile rank   16 tile number  32 net size 256
// tile8  rank in tile  7 tile size  8 tile rank    0 tile number   4 net size 32
// tile4  rank in tile  3 tile size  4 tile rank    1 tile number   2 net size 8

#include "hip/hip_cooperative_groups.h"
#include "cx.h"

namespace cg = cooperative_groups;

template <int T> __device__ void show_tile(const char *tag,cg::thread_block_tile<T> p)
{
	int rank =  p.thread_rank();     // thread rank in tile
	int size =  p.size();            // number of threads in tile
	int mrank = p.meta_group_rank(); // rank of tile in parent
	int msize = p.meta_group_size(); // number of tiles in parent

	printf("%s rank in tile %2d tile size %2d tile rank  %3d tile number %3d net size %d\n", 
		        tag, rank, size, mrank, msize, msize*size);
}
__global__ void cgwarp(int id)
{
	auto grid    = cg::this_grid();          // standard cg
	auto block   = cg::this_thread_block();  // definitions
	auto warp32  = cg::tiled_partition<32>(block); // 32 thread warps on block
	auto warp16  = cg::tiled_partition<16>(block); // 16 thread tiles on block
	auto warp8  = cg::tiled_partition< 8>(block);  //  8 thread tiles on block
	auto tile8 = cg::tiled_partition<8>(warp32); //  8 thread tiles on warp32
	auto tile4 = cg::tiled_partition<4>(tile8);  //  4 thread tiles on tile8
	if(grid.thread_rank() == id) {
		printf("warps and subwarps for thread %d:\n",id);
		show_tile<32>("warp32",warp32);
		show_tile<16>("warp16",warp16);
		show_tile< 8>("warp8 ",warp8);
		show_tile< 8>("tile8 ",tile8);
		show_tile< 4>("tile4 ",tile4);
	}
}
int main(int argc,char *argv[])
{
	int id      = (argc > 1) ? atoi(argv[1]) : 12345;
	int blocks  = (argc > 2) ? atoi(argv[2]) : 28800;
	int threads = (argc > 3) ? atoi(argv[3]) : 256;
	cgwarp<<<blocks,threads>>>(id);
    hipDeviceSynchronize(); // necessary in Linux to see kernel printf
	return 0;
}
