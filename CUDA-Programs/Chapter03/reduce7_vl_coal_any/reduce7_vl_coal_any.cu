#include "hip/hip_runtime.h"
// Programming in Parallel with CUDA - supporting code by Richard Ansorge 
// copyright 2021 is licensed under CC BY-NC 4.0 for non-commercial use
// This code may be freely changed but please retain an acknowledgement

// reduce7_vl_coal_any example 3.11 
// RTX 2070
// C:\bin\reduce7_vl_coal_any.exe 26 256 256 1000
// sum of 67108864 numbers: host 33557315.6 55.240 ms GPU 33557372928.0 0.927 ms
// 
// RTX 3080
// C:\bin\reduce7_vl_coal_any.exe 26 256 256  1000
// sum of 67108864 numbers: host 33557315.6 63.886 ms GPU 33557372928.0 0.537 ms

#include "hip/hip_cooperative_groups.h"
#include "cx.h"
#include "cxtimers.h"
#include <random>
#include "hip/hip_vector_types.h"

namespace cg = cooperative_groups;


__global__ void reduce7_vl(r_Ptr<float> sums,cr_Ptr<float> data,int n)
{
	// This kernel assumes the array sums is set to zeros on entry
	// and n is a multiple of 4.
	auto grid =  cg::this_grid();
	auto block = cg::this_thread_block();
	auto warp =  cg::tiled_partition<32>(block);
	float4 v4 ={0.0f,0.0f,0.0f,0.0f};  // use v4 to read global memory
	for(int tid = grid.thread_rank(); tid < n/4; tid += grid.size())
		v4 += reinterpret_cast<const float4 *>(data)[tid];
	float v =  v4.x + v4.y + v4.z + v4.w;  // accumulate thread sums in v
	warp.sync();
	v += warp.shfl_down(v,16); // |
	v += warp.shfl_down(v,8);  // | warp level
	v += warp.shfl_down(v,4);  // | reduce here
	v += warp.shfl_down(v,2);  // |
	v += warp.shfl_down(v,1);  // |
							   //     use atomicAdd to sum over warps
	if(warp.thread_rank()==0) atomicAdd(&sums[block.group_index().x],v);
}

__device__ void reduce7_vl_coal_any(r_Ptr<float>sums,cr_Ptr<float>data,int n)
{
	// This function works for any value of a.size() in [1,32] 
	// it assumes that n is a multiple of 4
	auto g = cg::this_grid();
	auto b = cg::this_thread_block();
	auto w = cg::tiled_partition<32>(b); // whole warp
	auto a = cg::coalesced_threads();    // active threads in warp
	int warps = g.group_dim().x*w.meta_group_size(); // number of warps in grid
	// divide data into contiguous parts, with one part per warp 
	int part_size = ((n/4)+warps-1)/warps;
	int part_start = (b.group_index().x*w.meta_group_size() +
		w.meta_group_rank())*part_size;
	int part_end = min(part_start+part_size,n/4);
	// get part sub-sums into threads of a
	float4 v4 ={0,0,0,0};
	int id = a.thread_rank();
	for(int k=part_start+id; k<part_end; k+=a.size()) // adjacent adds within
		v4 += reinterpret_cast<const float4 *>(data)[k]; //    the warp
	float v = v4.x + v4.y + v4.z + v4.w;
	a.sync();
	// now reduce over a
	// first deal with items held by ranks >= kstart
	int kstart = 1 << (31 - __clz(a.size())); // max power of 2 <= a.size()
	if(a.size() > kstart) {
		float w = a.shfl_down(v,kstart);
		if(a.thread_rank() < a.size()-kstart) v += w;// only update v for         
		a.sync();                                    // valid low ranking threads
	}
	// then do power of 2 reduction
	for(int k = kstart/2; k>0; k /= 2) v += a.shfl_down(v,k);
	if(a.thread_rank() == 0) atomicAdd(&sums[b.group_index().x],v);
}

__global__ void reduce7_any(r_Ptr<float>sums,cr_Ptr<float>data,int n)
{
	if(threadIdx.x % 3 == 0)  reduce7_vl_coal_any(sums,data,n);
}


int main(int argc,char *argv[])
{
	int N       = (argc > 1) ? 1 << atoi(argv[1]) : 1 << 24; // default 2^24
	int blocks  = (argc > 2) ? atoi(argv[2]) : 256;
	int threads = (argc > 3) ? atoi(argv[3]) : 256;  // multiple of 32
	int nreps   = (argc > 4) ? atoi(argv[4]) : 1000; // set this to 1 for correct answer or >> 1 for timing tests
	thrust::host_vector<float>    x(N);
	thrust::device_vector<float>  dx(N);
	thrust::device_vector<float>  dy(blocks);  // only even elements are used

	// initialise x with random numbers and copy to dx.
	std::default_random_engine gen(12345678);
	std::uniform_real_distribution<float> fran(0.0,1.0);
	for(int k = 0; k<N; k++) x[k] = fran(gen);
	dx = x;  // H2D copy (N words)
	cx::timer tim;
	double host_sum = 0.0;
	for(int k = 0; k<N; k++) host_sum += x[k]; // host reduce!
	double t1 = tim.lap_ms();

	tim.reset();
	// NB tacit assumtion that output array preset to zero. This is only needed to get correct result
	// for case nreps=1. Larger values of nreps are only used for timing purposes.	
	for(int rep=0;rep<nreps;rep++){
		reduce7_any<<<blocks,threads>>>(dy.data().get(),dx.data().get(),N);
	}

	// use reduce7_vl for final step
	dx[0] = 0.0f; // clear output buffer
	reduce7_vl<<<1,blocks>>>(dx.data().get(),dy.data().get(),blocks);
	hipDeviceSynchronize();
	double t2 = tim.lap_ms()/nreps;

	double gpu_sum = dx[0];  // D2H copy (1 word)
	printf("sum of %d numbers: host %.1f %.3f ms GPU %.1f %.3f ms\n",N,host_sum,t1,gpu_sum,t2);
	return 0;
}
