#include "hip/hip_runtime.h"
// Programming in Parallel with CUDA - supporting code by Richard Ansorge 
// copyright 2021 is licensed under CC BY-NC 4.0 for non-commercial use
// This code may be freely changed but please retain an acknowledgement

// program memtests includes examples 9.3 - 9.8 
// 
// C:\bin\memtests.exe 0..7 256 256 24
//
// RTX 2070
// test 0 total time 207.968 kernel time 11.975 ms
// test 1 total time 203.001 kernel time 5.661 ms
// test 2 total time 214.865 kernel time 12.058 ms
// test 3 total time 207.367 kernel time 5.649 ms
// test 4 total time 222.829 kernel time 5.646 ms
// test 5 total time 215.511 kernel time 5.752 ms
// test 6 total time 782.523 kernel time 60.889 ms
// test 7 total time 755.528 kernel time 67.476 ms
//
// RTX 3080
// test 0 total time 123.757 kernel time 8.076 ms
// test 1 total time 144.661 kernel time 5.939 ms
// test 2 total time 131.650 kernel time 8.045 ms
// test 3 total time 164.538 kernel time 5.926 ms
// test 4 total time 140.808 kernel time 5.964 ms
// test 5 total time 155.775 kernel time 5.826 ms
// test 6 total time 631.557 kernel time 50.210 ms
// test 7 total time 629.506 kernel time 49.840 ms
// 
// RTX 3080 Linux note tests 6 & 7 much improved
// test 0 total time 180.471 kernel time 7.095 ms
// test 1 total time 232.240 kernel time 5.586 ms
// test 2 total time 363.982 kernel time 7.407 ms
// test 3 total time 398.766 kernel time 5.548 ms
// test 4 total time 364.661 kernel time 5.570 ms
// test 5 total time 190.071 kernel time 5.586 ms
// test 6 total time 195.511 kernel time 21.775 ms
// test 7 total time 190.013 kernel time 6.196 ms
//
// NB for the uint data type used here the reduction step will overflow for
// buffer sizes greater than about 2^24. The timing information remains correct

#include "hip/hip_cooperative_groups.h"
#include "cx.h"
#include "cxtimers.h"
#include "hip/hip_vector_types.h"

namespace cg = cooperative_groups;

// best reduce version 
__global__ void reduce_warp_vl(r_Ptr<uint> sums,cr_Ptr<uint> data,uint n)
{
	auto b = cg::this_thread_block();    // thread block
	auto w = cg::tiled_partition<32>(b); // warp

	int4 v4 ={0,0,0,0};
	for(int tid = b.size()*b.group_index().x+b.thread_rank(); tid < n/4;
		tid += b.size()*gridDim.x) v4 += reinterpret_cast<const int4 *>(data)[tid];

	uint v = v4.x + v4.y + v4.z + v4.w;
	w.sync();

	v += w.shfl_down(v,16);
	v += w.shfl_down(v,8);
	v += w.shfl_down(v,4);
	v += w.shfl_down(v,2);
	v += w.shfl_down(v,1);
	if(w.thread_rank() == 0) atomicAdd(&sums[b.group_index().x],v);
}

double fill_buf(uint *buf,uint dsize)
{
	double sum = 0.0;
	for(uint k=0;k<dsize;k++) {
		buf[k] = k%419;  // just test data
		sum += buf[k];   // host sum to check correctness
	}
	return sum;
}

// (A) classic CUDA version  exaple 9.4
int reduce_classic(int blocks,int threads,uint dsize,double &t)
{
	uint *host_buf = (uint *)malloc(dsize*sizeof(uint));     // full data host
	uint *dev_buf; hipMalloc(&dev_buf,dsize*sizeof(uint));  // full data device
	uint *dev_sum; hipMalloc(&dev_sum,blocks*sizeof(uint)); // block sums(device only)
	uint host_tot;                                           // final sum host (scalar)
	uint *dev_tot; hipMalloc(&dev_tot,1*sizeof(uint));      // final sum device

	double check = fill_buf(host_buf,dsize);
	cx::timer cuda;

	hipMemcpy(dev_buf,host_buf,dsize*sizeof(uint),hipMemcpyHostToDevice);
	reduce_warp_vl<<<blocks,threads>>>(dev_sum,dev_buf,dsize);
	reduce_warp_vl<<<     1,blocks>>>(dev_tot,dev_sum,blocks);
	hipMemcpy(&host_tot,dev_tot,sizeof(uint),hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	t = cuda.lap_ms();
	if(check != host_tot) printf("error classic: sum %u check %.0f\n",host_tot,check);

	free(host_buf);
	hipFree(dev_buf);
	hipFree(dev_sum);
	hipFree(dev_tot);

	return 0;
}

// (B) CUDA classic with host pinned memory  example 9.5
int reduce_classic_pinned(int blocks,int threads,uint dsize,double &t)
{
	uint *host_buf; hipHostMalloc(&host_buf,dsize*sizeof(uint)); // full data host
	uint *dev_buf;  hipMalloc(&dev_buf,dsize*sizeof(uint));      // full data device
	uint *dev_sum;  hipMalloc(&dev_sum,blocks*sizeof(uint));     // block sums(device only)

	uint host_tot;                                             // final sum host
	uint *dev_tot; hipMalloc(&dev_tot,1*sizeof(uint));  // final sum device

	double check = fill_buf(host_buf,dsize);
	cx::timer cuda;
	hipMemcpy(dev_buf,host_buf,dsize*sizeof(uint),hipMemcpyDefault);
	reduce_warp_vl<<<blocks,threads>>>(dev_sum,dev_buf,dsize);
	reduce_warp_vl<<<     1,blocks>>>(dev_tot,dev_sum,blocks);
	hipMemcpy(&host_tot,dev_tot,sizeof(uint),hipMemcpyDefault);
	hipDeviceSynchronize();
	t = cuda.lap_ms();

	if(check != host_tot) printf("error classic pinned: sum %u check %.0f\n",host_tot,check);

	hipHostFree(host_buf);
	hipFree(dev_buf);
	hipFree(dev_sum);
	hipFree(dev_tot);

	return 0;
}

// (C) thrust container standard host memory (not in book)
int reduce_thrust_standard(int blocks,int threads,uint dsize,double &t)
{
	thrustHvec<uint> host_buf(dsize);  // full data host
	thrustDvec<uint> dev_buf(dsize);  // full data device
	thrustDvec<uint> dev_sum(blocks); // block sums (device only)
	thrustHvec<uint> host_tot(1);     // final sum host
	thrustDvec<uint> dev_tot(1);      // final sum device

	double check = fill_buf(host_buf.data(),dsize);
	cx::timer cuda;
	dev_buf = host_buf;
	reduce_warp_vl<<<blocks,threads>>>(dev_sum.data().get(),dev_buf.data().get(),dsize);
	reduce_warp_vl<<<     1,blocks>>>(dev_tot.data().get(),dev_sum.data().get(),blocks);
	host_tot = dev_tot;
	hipDeviceSynchronize();
	t = cuda.lap_ms();

	if(check != host_tot[0]) printf("error normal reduce standard done sum %u check %.0f\n",host_tot[0],check);

	return 0;
}

// (D) thust container with pinned host memory example 9.6
int reduce_thrust_pinned(int blocks,int threads,uint dsize,double &t)
{
	thrustHvecPin<uint> host_buf(dsize);  // full data host
	thrustDvec<uint>     dev_buf(dsize);  // full data device
	thrustDvec<uint>     dev_sum(blocks); // block sums (device only)
	thrustHvecPin<uint>  host_tot(1);     // final sum host
	thrustDvec<uint>     dev_tot(1);      // final sum device

	double check = fill_buf(host_buf.data(),dsize);
	cx::timer cuda;
	dev_buf = host_buf;
	reduce_warp_vl<<<blocks,threads>>>(dev_sum.data().get(),dev_buf.data().get(),dsize);
	reduce_warp_vl<<<     1,blocks>>>(dev_tot.data().get(),dev_sum.data().get(),blocks);
	host_tot = dev_tot;
	hipDeviceSynchronize();
	t = cuda.lap_ms();

	if(check != host_tot[0]) printf("error normal reduce done sum %u check %.0f\n",host_tot[0],check);

	return 0;
}

// (E) thrust container with memcpy hybrid (not in book)
int reduce_thrust_hybrid(int blocks,int threads,uint dsize,double &t)
{
	thrustHvecPin<uint> host_buf(dsize); // full data host
	thrustDvec<uint> dev_buf(dsize);  // full data device
	thrustDvec<uint> dev_sum(blocks); // block sums (device only)
	thrustHvecPin<uint> host_tot(1);     // final sum host
	thrustDvec<uint> dev_tot(1);      // final sum device

	double check = fill_buf(host_buf.data(),dsize);
	cx::timer cuda;
	hipMemcpy(dev_buf.data().get(),host_buf.data(),dsize*sizeof(uint),hipMemcpyDefault);
	reduce_warp_vl<<<blocks,threads>>>(dev_sum.data().get(),dev_buf.data().get(),dsize);
	reduce_warp_vl<<<     1,blocks>>>(dev_tot.data().get(),dev_sum.data().get(),blocks);
	hipMemcpy(host_tot.data(),dev_tot.data().get(),sizeof(uint),hipMemcpyDefault);
	hipDeviceSynchronize();
	t = cuda.lap_ms();

	if(check != host_tot[0]) printf("error reduce hybrid done sum %u check %.0f\n",host_tot[0],check);

	return 0;
}

// (F) zero-copy/mapped memory version example 9.7
int reduce_zerocopy(int blocks,int threads,uint dsize,double &t)
{
	uint *host_buf; hipHostAlloc(&host_buf,dsize*sizeof(uint),hipHostMallocMapped);
	uint *host_sum; hipHostAlloc(&host_sum,blocks*sizeof(uint),hipHostMallocMapped);
	uint *host_tot; hipHostAlloc(&host_tot,1*sizeof(uint),hipHostMallocMapped);

	uint *dev_buf; hipHostGetDevicePointer(&dev_buf,host_buf,0);
	uint *dev_sum; hipHostGetDevicePointer(&dev_sum,host_sum,0);
	uint *dev_tot; hipHostGetDevicePointer(&dev_tot,host_tot,0);


	double check = fill_buf(host_buf,dsize);
	cx::timer cuda;

	reduce_warp_vl<<<blocks,threads>>>(dev_sum,dev_buf,dsize);
	reduce_warp_vl<<<     1,blocks>>>(dev_tot,dev_sum,blocks);
	hipDeviceSynchronize();
	t = cuda.lap_ms();

	if(check != host_tot[0]) printf("error  mapped: sum %u check %.0f\n",host_tot[0],check);

	hipHostFree(host_buf);
	hipHostFree(host_sum);
	hipHostFree(host_tot);

	return 0;
}

// (G) Managed Memory Version example 9.8
int reduce_managed(int blocks,int threads,uint dsize,double &t)
{
	uint *buf; hipMallocManaged(&buf,dsize*sizeof(uint));  // full data
	uint *sum; hipMallocManaged(&sum,blocks*sizeof(uint));  // block sums
	uint *tot; hipMallocManaged(&tot,sizeof(uint));  // grand total

	double check = fill_buf(buf,dsize);
	cx::timer cuda;
	reduce_warp_vl<<<blocks,threads>>>(sum,buf,dsize);
	reduce_warp_vl<<<     1,blocks>>>(tot,sum,blocks);
	hipDeviceSynchronize(); // necessary
	t = cuda.lap_ms();

	if(check != tot[0]) printf("error managed: sum %u check %.0f\n",tot[0],check);

	hipFree(sum);
	hipFree(buf);
	hipFree(tot);

	return 0;
}

// (H) advanced managed version requires Linux Driver for proper test (not in book)
int reduce_advanced_managed(int blocks,int threads,uint dsize,double &t)
{
	uint *buf; hipMallocManaged(&buf,dsize*sizeof(uint),hipMemAttachHost);  // full data
	uint *sum; hipMallocManaged(&sum,blocks*sizeof(uint));  // block sums
	uint *tot; hipMallocManaged(&tot,sizeof(uint));  // grand total

	hipStream_t s1; hipStreamCreate(&s1);

	double check = fill_buf(buf,dsize);
	hipStreamAttachMemAsync(s1,buf);
	hipDeviceSynchronize();
	cx::timer cuda;
	reduce_warp_vl<<<blocks,threads,0,s1>>>(sum,buf,dsize);
	reduce_warp_vl<<<     1,blocks>>>(tot,sum,blocks);
	hipDeviceSynchronize(); // necessary
	t = cuda.lap_ms();

	if(check != tot[0]) printf("error unified advance reduce done sum %u check %.0f\n",tot[0],check);

	hipStreamDestroy(s1);
	hipFree(sum);
	hipFree(buf);
	hipFree(tot);

	return 0;
}

// this version has better printing than book
int main(int argc,char *argv[])
{
	if(argc < 2){
		printf("usage: memtests.exe <test|0> <blocks|256> <threads|256> <size as power of 2|24>\n\n");
		printf("test = 0: classic          using malloc for host and hipMalloc for device\n");
		printf("test = 1: classic_pinned   using hipHostMalloc for host and cudaMallocfor device\n");
		printf("test = 2: thrust_standard  using thrust for host and device\n");
		printf("test = 3: thrust_pinned    as 2 but using pinned host memory\n");
		printf("test = 4: thrust_hybrid    as 3 but using hipMemcpy instead of thrust copy\n");
		printf("test = 5: zerocopy         using hipHostAlloc for both host and device\n");
		printf("test = 6: managed          using hipMallocManaged for both host and device\n");
		printf("test = 7: advanced_managed as 6 but with hipStreamAttachMemAsync\n");
		return 0;
	}

	std::vector<std::string> tag(8);
	tag[0] = "classic";
	tag[1] = "classic pinned";
	tag[2] = "thrust";
	tag[3] = "thrust pinned";
	tag[4] = "thrust hydrid";
	tag[5] = "zero-copy/managed";
	tag[6] = "managed";
	tag[7] = "advanced managed";

	int unify  =  (argc > 1) ? atoi(argv[1]) : 0;
	int blocks =  (argc > 2) ? atoi(argv[2]) : 256;
	int threads = (argc > 3) ? atoi(argv[3]) : 256;
	uint dsize =  (argc > 4) ? 1 << atoi(argv[4]) : 1 << 24;
	if(dsize > 16777216) printf("Warning dsize = %u, error likly if dsize > 2^24, timing results still correct\n",dsize);

	double t2 = 0.0;
	cx::timer tim;
	if     (unify==0) reduce_classic(blocks,threads,dsize,t2);
	else if(unify==1) reduce_classic_pinned(blocks,threads,dsize,t2);
	else if(unify==2) reduce_thrust_standard(blocks,threads,dsize,t2);
	else if(unify==3) reduce_thrust_pinned(blocks,threads,dsize,t2);
	else if(unify==4) reduce_thrust_hybrid(blocks,threads,dsize,t2);
	else if(unify==5) reduce_zerocopy(blocks,threads,dsize,t2);
	else if(unify==6) reduce_managed(blocks,threads,dsize,t2);
	else if(unify==7) reduce_advanced_managed(blocks,threads,dsize,t2);
	else { printf("unify must be in range 0-7 got %d\n",unify); return 1; }

	double t1 = tim.lap_ms();
	//printf("test %s total time %.3f kernel time %.3f ms\n",tag[unify].c_str(),t1,t2);
	printf("test %d total time %.3f kernel time %.3f ms\n",unify,t1,t2);

	FILE *flog = fopen("unify.txt","a");
	int a4 = (argc > 4) ? atoi(argv[4]) : 24;
	fprintf(flog,"%2d %2d %.3f %.3f\n",unify,a4,t1,t2);
	fclose(flog);

	std::atexit([]{hipDeviceReset();});

	return 0;
}

// version of main in book (not used here)
int main_book(int argc,char *argv[])
{
	int test  =   (argc > 1) ? atoi(argv[1]) : 0;
	int blocks =  (argc > 2) ? atoi(argv[2]) : 256;
	int threads = (argc > 3) ? atoi(argv[3]) : 256;
	uint dsize =  (argc > 4) ? 1 << atoi(argv[4]) : 1 << 24;
	double t2 = 0.0;  // kernel time

	cx::timer tim;
	if(test==0)      reduce_classic(blocks,threads,dsize,t2);
	else if(test==1) reduce_classic_pinned(blocks,threads,dsize,t2);
	else if(test==2) reduce_thrust_standard(blocks,threads,dsize,t2);
	else if(test==3) reduce_thrust_pinned(blocks,threads,dsize,t2);
	else if(test==4) reduce_thrust_hybrid(blocks,threads,dsize,t2);
	else if(test==5) reduce_zerocopy(blocks,threads,dsize,t2);
	else if(test==6) reduce_managed(blocks,threads,dsize,t2);
	else  return 1;

	double t1 = tim.lap_ms();
	printf("test %d total time %.3f kernel time %.3f ms\n",test,t1,t2);

	std::atexit([]{hipDeviceReset();});
	return 0;
}

