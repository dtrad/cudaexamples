// Programming in Parallel with CUDA - supporting code by Richard Ansorge 
// copyright 2021 is licensed under CC BY-NC 4.0 for non-commercial use
// This code may be freely changed but please retain an acknowledgement

// example 6.6 piH6 cuRand Host API hipMemcpyAsync

#include "cx.h"  
#include "cxtimers.h"
#include "hiprand.h"
#include <random>

void sum_part(cr_Ptr<float> rnum,int tries,long long &pisum)
{
	unsigned int sum = 0;
	for(int i=0;i<tries;i++){
		float x = rnum[i*2];
		float y = rnum[i*2+1];
		if(x*x + y*y < 1.0f) sum++;
	}
	pisum += sum;
}

int main(int argc,char *argv[])
{
	std::random_device rd;
	int points = 1000000;
	int passes =        (argc > 1) ? atoi(argv[1]) : 1;
	unsigned int seed = (argc > 2) ? atoi(argv[2]) : rd();

	long long pisum = 0;

	int bsize = points*2*sizeof(float);
	float *a;       hipHostMalloc(&a,bsize);   // host buffers a and b
	float *b;       hipHostMalloc(&b,bsize);   // in pinned memory
	float *dev_rdm; hipMalloc(&dev_rdm,bsize); // single device buffer

	hipEvent_t copydone; hipEventCreate(&copydone);  // CUDA event

	cx::timer tim;   // overall time

	hiprandGenerator_t gen;
	hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_DEFAULT);

	//hiprandSetGeneratorOrdering(gen,HIPRAND_ORDERING_PSEUDO_SEEDED);
	hiprandSetPseudoRandomGeneratorSeed(gen,seed);
	hiprandGenerateUniform(gen,dev_rdm,points*2);
	hipMemcpy(a,dev_rdm,bsize,hipMemcpyDeviceToHost);  // get 1st block in a 

	for(int k = 0; k < passes; k++) {
		hiprandGenerateUniform(gen,dev_rdm,points*2);
		hipMemcpyAsync(b,dev_rdm,bsize,hipMemcpyDeviceToHost); // async copy to b
		hipEventRecord(copydone,0);
		hipEventQuery(copydone);  // WHY DO I NEED THIS event with streams???????
		sum_part(a,points,pisum);    //  process a while b downloading
		std::swap(a,b);
		hipStreamWaitEvent(0,copydone,0);
	}
	double t1 = tim.lap_ms();

	double pi = 4.0*(double)pisum / ((double)points*(double)passes);
	long long ntot = passes*points;
	double frac_error = 1000000.0*(pi - cx::pi<double>)/cx::pi<double>; // error ppm
	printf("pi = %10.8f err %.1f, ntot %lld, time %.3f ms\n",pi,frac_error,ntot,t1);

	// tidy up
	hipHostFree(a); hipHostFree(b); hipFree(dev_rdm);
	hiprandDestroyGenerator(gen);
	return 0;
}
