#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
#include <iostream>
// compile with nvcc -O2 -arch=sm_75 cudafftex.cu -o cudafftex -lcufft
// note: original chatGPT did not include a complex device data 
// so it did not compile since R2C requires a complex vector.
// Also added plan2 to go back.

int main() {
  // Allocate host and device arrays
  const int N = 8;
  float h_data[N] = {1, 2, 3, 4, 5, 6, 7, 8};
  float* d_data;
  hipfftComplex* d_cdata;

  hipMalloc((void**)&d_data, N * sizeof(float));
  hipMalloc((void **)&d_cdata,sizeof(hipfftComplex)*N );
  // Copy data from host to device
  hipMemcpy(d_data, h_data, N * sizeof(float), hipMemcpyHostToDevice);

  // Set up the FFT plan
  hipfftHandle plan1;
  hipfftHandle plan2;
  hipfftPlan1d(&plan1, N, HIPFFT_R2C, 1);
  hipfftPlan1d(&plan2, N, HIPFFT_C2R, 0);

  // Execute the FFT
  hipfftExecR2C(plan1, d_data, d_cdata);
  hipfftExecC2R(plan2, d_cdata, d_data);  
  // Copy the result back to the host
  hipMemcpy(h_data, d_data, N * sizeof(float), hipMemcpyDeviceToHost);

  // Print the result
  std::cout << "FFT result: ";
  for (int i = 0; i < N; i++) {
    std::cout << h_data[i] << " ";
  }
  std::cout << std::endl;

  // Clean up
  hipfftDestroy(plan1);
  hipfftDestroy(plan2);
  hipFree(d_data);
  hipFree(d_cdata);

  return 0;
}
