#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void someKernel(float* data, unsigned int size){
    int i= blockIdx.x*blockDim.x + threadIdx.x;
    if (i<size) data[i]=1;
}

void initarray(float* data, unsigned int size){
    for (int i=0;i<size;i++) data[i]=i;
}
int main() {
  unsigned int sizeint = 1<<23;
  unsigned int size=sizeint*sizeof(float);

  // Create events for timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Allocate pinned memory on the host (CPU)
  float *h_data = (float*) malloc(size);

  // Transfer data from the host to the device (GPU)
  float *d_data;
  hipMalloc((void**)&d_data, size);
  initarray(h_data,sizeint);
  hipMemcpy(d_data, h_data, size, hipMemcpyHostToDevice);



  // Record the start time
  hipEventRecord(start, 0);

  // Use the data on the device
  someKernel<<<1,1>>>(d_data,sizeint);

  // Record the stop time and wait for the kernel to finish
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);


  // Transfer data back from the device to the host
  hipMemcpy(h_data, d_data, size, hipMemcpyDeviceToHost);

  // Free device and host memory
  hipFree(d_data);
  free(h_data);

  // Calculate the elapsed time
  float elapsed_time;
  hipEventElapsedTime(&elapsed_time, start, stop);
  printf("Elapsed time: %f ms\n", elapsed_time);



  // Destroy the events
  hipEventDestroy(start);
  hipEventDestroy(stop);
  
  return 0;
}

