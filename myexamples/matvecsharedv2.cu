#include <iostream>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 8
using namespace std;
void multiply(float* A, const float* x, float* y, int M, int N){
    for (int i = 0; i < M; ++i) {
        for (int j = 0; j < N; ++j) {
            y[i] += A[i*N+j] * x[j];
        }
    }
    return;
}

 
__global__ void matvec_kernel(float *A, float *x, float *y, int M, int N) {
    // Determine the thread's row and column within the block
    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;

    // Allocate shared memory for the block
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float xs[BLOCK_SIZE];

    float result = 0.0f;
    // need a loop here for the different tiles
    for (int ph=0; ph < N/BLOCK_SIZE; ++ph){
        // Load the element of A and x into shared memory
        //As[threadIdx.y][threadIdx.x] = A[ph*BLOCK_SIZE + row * N + col];
        //xs[threadIdx.x] = x[ph*BLOCK_SIZE+col];

        As[threadIdx.y][threadIdx.x] = (row < M && col < N) ? A[ph*BLOCK_SIZE + row * N + threadIdx.x] : 0.0f;
        xs[threadIdx.x] = (col < N) ? x[ph*BLOCK_SIZE+threadIdx.x] : 0.0f; 
        __syncthreads();

        // Perform the dot product of the row of A and x
        
        for (int i = 0; i < BLOCK_SIZE; i++) {
            result += As[threadIdx.y][i] * xs[i];
        }
        __syncthreads();
    }
        // Store the result in the output vector y
    y[col*M+row] = result;
        
    
}

int main() {
    // Allocate host and device arrays
    const int m = 150;
    int n = 65;
    int nbsize=BLOCK_SIZE;
    int norig=n;
    while (n%nbsize) n++;
    cout << "n=" << n << endl;
    float* h_A =(float*) malloc(m*n*sizeof(float));
    float* h_x =(float*) malloc(n*sizeof(float));
    float* h_y  =(float*) malloc(m*sizeof(float));
    float* h_y2 =(float*) malloc(m*sizeof(float));

    for (int i=0;i<m;i++) for (int j=0;j<n;j++) h_A[i*n+j]=0;
    for (int i=0;i<m;i++) for (int j=0;j<norig;j++) h_A[i*n+j]=i+j;
    for (int i=0;i<n;i++) h_x[i]=0;    
    for (int i=0;i<norig;i++) h_x[i]=i;
    for (int i=0;i<m;i++) h_y[i]=0;
    for (int i=0;i<m;i++) h_y2[i]=0;
    multiply(h_A, h_x, h_y, m, n);

    // Print the result
    std::cout << "CPU Result: ";
    for (int i = 0; i < m; i++) std::cout << h_y[i] << " ";
    std::cout << std::endl;

    float* d_A=0;
    float* d_x=0;
    float* d_y=0;
    hipMalloc((void**)&d_A, m * n * sizeof(float));
    hipMalloc((void**)&d_x, n * sizeof(float));
    hipMalloc((void**)&d_y, m * sizeof(float));
    

    // Copy data from host to device
    hipMemcpy(d_A, h_A, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, n * sizeof(float), hipMemcpyHostToDevice);
  
    
    // Launch the kernel
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (m + blockSize.y - 1) / blockSize.y);
    
    matvec_kernel<<< gridSize, blockSize >>>(d_A, d_x, d_y, m, n);
    hipMemcpy(h_y2, d_y, m * sizeof(float), hipMemcpyDeviceToHost);
     
    // Print the result

    std::cout << "GPU Result: ";
    for (int i = 0; i < m; i++) std::cout << h_y2[i] << " ";
    std::cout << std::endl;
    std::cout << "<<< (" << gridSize.x << ", " << gridSize.y << ")" ;
    std::cout << ",(" << blockSize.x << ", " << blockSize.y << ")" << ">>> " << std::endl;
  
    // difference
    float diff=0;
    for (int i = 0; i < m; i++) diff+=fabs(h_y[i]-h_y2[i]);
    std::cout << "diff " << diff << std::endl;

    // Clean up
    hipFree(d_A);
    hipFree(d_x);
    hipFree(d_y);
    free(h_A);
    free(h_x);
    free(h_y);
    free(h_y2);

    return 0;
  }
  