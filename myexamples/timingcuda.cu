#include <hip/hip_runtime.h>

int main() {
  // Create events for timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Record the start time
  hipEventRecord(start, 0);

  // Launch the kernel
  someKernel<<<1,1>>>();

  // Record the stop time and wait for the kernel to finish
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);

  // Calculate the elapsed time
  float elapsed_time;
  hipEventElapsedTime(&elapsed_time, start, stop);
  printf("Elapsed time: %f ms\n", elapsed_time);

  // Destroy the events
  hipEventDestroy(start);
  hipEventDestroy(stop);

  return 0;
}
